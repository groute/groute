#include "hip/hip_runtime.h"
// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its 
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
#include <vector>
#include <algorithm>
#include <thread>
#include <memory>
#include <random>

#include <gflags/gflags.h>

#include <groute/event_pool.h>
#include <groute/fused_distributed_worklist.h>
#include <groute/fused_worker.h>
#include <groute/cta_work.h>

#include <groute/graphs/csr_graph.h>
#include <groute/graphs/traversal_algo.h>
#include <groute/graphs/fused_solver.h>

#include <utils/parser.h>
#include <utils/utils.h>
#include <utils/stopwatch.h>
#include <utils/markers.h>

#include "bfs_common.h"

DECLARE_int32(source_node);

DEFINE_bool(exitonerror, false, "exit on error");


namespace bfs {
    namespace opt {
        
        const level_t INF = UINT_MAX;

        struct LevelData
        {
            index_t node;
            level_t level;

            __device__ __host__ __forceinline__ LevelData(index_t node, level_t level) : node(node), level(level) { }
            __device__ __host__ __forceinline__ LevelData() : node(INF), level(INF) { }
        };

        typedef index_t local_work_t;
        typedef LevelData remote_work_t;

        __global__ void BFSInit(level_t* levels, int nnodes)
        {
            int tid = GTID;
            if (tid < nnodes)
            {
                levels[tid] = INF;
            }
        }

        template<
            typename TGraph,
            typename TGraphDatum>
        struct BFSWorkNP
        {
            template<typename WorkSource>
            __device__ static void work(
                const WorkSource& work_source,
                groute::dev::CircularWorklist<local_work_t>& rwl_in,
                groute::dev::CircularWorklist<remote_work_t>& rwl_out,
                const TGraph& graph, TGraphDatum& levels_datum
                )
            {
                uint32_t tid = TID_1D;
                uint32_t nthreads = TOTAL_THREADS_1D;

                uint32_t work_size = work_source.get_size();
                uint32_t work_size_rup = round_up(work_size, blockDim.x) * blockDim.x; // we want all threads in active blocks to enter the loop

                for (uint32_t i = 0 + tid; i < work_size_rup; i += nthreads)
                {
                    groute::dev::np_local<level_t> np_local = { 0, 0, 0 };

                    if (i < work_size)
                    {
                        index_t node = work_source.get_work(i);
                        np_local.start = graph.begin_edge(node);
                        np_local.size = graph.end_edge(node) - np_local.start;
                        np_local.meta_data = levels_datum.get_item(node) + 1;
                    }

                    groute::dev::CTAWorkScheduler<level_t>::template schedule(
                        np_local, 
                        [&graph, &levels_datum, &rwl_in, &rwl_out](index_t edge, level_t next_level)
                        {
                            index_t dest = graph.edge_dest(edge);
                            if (next_level < atomicMin(levels_datum.get_item_ptr(dest), next_level))
                            {
                                int is_owned = graph.owns(dest);

                                // TODO: move ballot logic to a device structure   

                                int owned_mask = __ballot_sync(__activemask(), is_owned ? 1 : 0);
                                int remote_mask = __ballot_sync(__activemask(), is_owned ? 0 : 1);

                                if (is_owned)
                                {
                                    int high_leader = __ffs(owned_mask) - 1;
                                    int thread_offset = __popc(owned_mask & ((1 << lane_id()) - 1));
                                    rwl_in.prepend_warp(dest, high_leader, __popc(owned_mask), thread_offset);
                                }
                                else
                                {
                                    int low_leader = __ffs(remote_mask) - 1;
                                    int thread_offset = __popc(remote_mask & ((1 << lane_id()) - 1));
                                    rwl_out.append_warp(LevelData(dest, next_level), low_leader, __popc(remote_mask), thread_offset);
                                }
                            }
                        }
                        ); 
                }
            }
        };

        template<
            typename TGraph,
            typename TGraphDatum>
        struct BFSWork
        {
            template<typename WorkSource>
            __device__ static void work(
                const WorkSource& work_source,
                groute::dev::CircularWorklist<local_work_t>& rwl_in,
                groute::dev::CircularWorklist<remote_work_t>& rwl_out,
                const TGraph& graph, TGraphDatum& levels_datum
                )
            {
                uint32_t tid = TID_1D;
                uint32_t nthreads = TOTAL_THREADS_1D;

                uint32_t work_size = work_source.get_size();

                for (uint32_t i = 0 + tid; i < work_size; i += nthreads)
                {
                    index_t node = work_source.get_work(i);
                    level_t next_level = levels_datum.get_item(node) + 1;

                    for (index_t edge = graph.begin_edge(node), end_edge = graph.end_edge(node); edge < end_edge; ++edge)
                    {
                        index_t dest = graph.edge_dest(edge);
                        if (next_level < atomicMin(levels_datum.get_item_ptr(dest), next_level))
                        {
                            int is_owned = graph.owns(dest);

                            // TODO: move ballot logic to a device structure   

                            int owned_mask = __ballot_sync(__activemask(), is_owned ? 1 : 0);
                            int remote_mask = __ballot_sync(__activemask(), is_owned ? 0 : 1);

                            if (is_owned)
                            {
                                int high_leader = __ffs(owned_mask) - 1;
                                int thread_offset = __popc(owned_mask & ((1 << lane_id()) - 1));
                                rwl_in.prepend_warp(dest, high_leader, __popc(owned_mask), thread_offset);
                            }
                            else
                            {
                                int low_leader = __ffs(remote_mask) - 1;
                                int thread_offset = __popc(remote_mask & ((1 << lane_id()) - 1));
                                rwl_out.append_warp(LevelData(dest, next_level), low_leader, __popc(remote_mask), thread_offset);
                            }
                        }
                    }
                }
            }
        };

        struct SplitOps
        {
        private:
            groute::graphs::dev::CSRGraphSeg m_graph_seg;
            groute::graphs::dev::GraphDatum<level_t> m_levels_datum;

        public:
            template<typename...UnusedData>
            SplitOps(const groute::graphs::dev::CSRGraphSeg& graph_seg, const groute::graphs::dev::GraphDatum<level_t>& levels_datum, UnusedData&... data)
                : m_graph_seg(graph_seg), m_levels_datum(levels_datum)
            {
            }

            __device__ __forceinline__ groute::opt::SplitFlags on_receive(const remote_work_t& work)
            {
                if (m_graph_seg.owns(work.node))
                {
                    return (work.level < atomicMin(m_levels_datum.get_item_ptr(work.node), work.level))
                        ? groute::opt::SF_Take
                        : groute::opt::SF_None; // filter
                }

                return groute::opt::SF_Pass;
            }

            __device__ __forceinline__ bool is_high_prio(const local_work_t& work, const level_t& global_prio)
            {
                return m_levels_datum[work] <= global_prio;
            }

            __device__ __forceinline__ groute::opt::SplitFlags on_send(local_work_t work)
            {
                return (m_graph_seg.owns(work))
                    ? groute::opt::SF_Take
                    : groute::opt::SF_Pass;
            }

            __device__ __forceinline__ remote_work_t pack(local_work_t work)
            {
                return LevelData(work, m_levels_datum.get_item(work));
            }

            __device__ __forceinline__ local_work_t unpack(const remote_work_t& work)
            {
                return work.node;
            }
        };

        template<typename TGraph, typename TGraphDatum>
        struct FusedProblem
        {
            TGraph m_graph;
            TGraphDatum m_levels_datum;

            typedef BFSWork<TGraph, TGraphDatum> WorkType;
            typedef BFSWorkNP<TGraph, TGraphDatum> WorkTypeNP;

        public:
            FusedProblem(const TGraph& graph, const TGraphDatum& levels_datum) :
                m_graph(graph), m_levels_datum(levels_datum)
            {
            }

            // Called before a global CPU+GPU barrier
            void Init(groute::Stream& stream) const
            {
                dim3 grid_dims, block_dims;
                KernelSizing(grid_dims, block_dims, m_levels_datum.size);

                BFSInit <<< grid_dims, block_dims, 0, stream.hip_stream >>>(
                    m_levels_datum.data_ptr, m_levels_datum.size);
            }

            bool DoFusedInit(groute::Worklist<local_work_t>* lwl_high, groute::Worklist<local_work_t>* lwl_low,
                groute::CircularWorklist<local_work_t>*  rwl_in, groute::CircularWorklist<remote_work_t>*  rwl_out,
                int fused_chunk_size, level_t global_prio,
                volatile int *high_work_counter, volatile int *low_work_counter,
                uint32_t *kernel_internal_counter, volatile int *send_signal_ptr,
                hipcub::GridBarrierLifetime& barrier_lifetime,
                dim3 grid_dims, dim3 block_dims, groute::Stream& stream)
            {
                return false; // no work was done here
            }

            void DoFusedWork(groute::Worklist<local_work_t>* lwl_high, groute::Worklist<local_work_t>* lwl_low,
                groute::CircularWorklist<local_work_t>*  rwl_in, groute::CircularWorklist<remote_work_t>*  rwl_out,
                int fused_chunk_size, level_t global_prio,
                volatile int *high_work_counter, volatile int *low_work_counter,
                uint32_t *kernel_internal_counter, volatile int *send_signal_ptr,
                hipcub::GridBarrierLifetime& barrier_lifetime,
                dim3 grid_dims, dim3 block_dims, groute::Stream& stream)
            {
                if (FLAGS_iteration_fusion)
                {
                    if (FLAGS_cta_np)
                    {
                        groute::FusedWork <
                            groute::NeverStop, local_work_t, remote_work_t, level_t, SplitOps,
                            WorkTypeNP,
                            TGraph, TGraphDatum >

                            <<< grid_dims, block_dims, 0, stream.hip_stream >>> (

                            lwl_high->DeviceObject(), lwl_low->DeviceObject(),
                            rwl_in->DeviceObject(), rwl_out->DeviceObject(),
                            fused_chunk_size, global_prio,
                            high_work_counter, low_work_counter,
                            kernel_internal_counter, send_signal_ptr,
                            barrier_lifetime,                       
                            bfs::opt::SplitOps(m_graph, m_levels_datum),
                            m_graph, m_levels_datum
                            );
                    }
                    else
                    {
                        groute::FusedWork <
                            groute::NeverStop, local_work_t, remote_work_t, level_t, SplitOps,
                            WorkType,
                            TGraph, TGraphDatum >

                            <<< grid_dims, block_dims, 0, stream.hip_stream >>> (

                            lwl_high->DeviceObject(), lwl_low->DeviceObject(),
                            rwl_in->DeviceObject(), rwl_out->DeviceObject(),
                            fused_chunk_size, global_prio,
                            high_work_counter, low_work_counter,
                            kernel_internal_counter, send_signal_ptr,
                            barrier_lifetime,
                            bfs::opt::SplitOps(m_graph, m_levels_datum),
                            m_graph, m_levels_datum
                            );
                    }
                }
                else
                {
                    if (FLAGS_cta_np)
                    {
                        groute::FusedWork <
                            groute::RunNTimes<1>, local_work_t, remote_work_t, level_t, SplitOps,
                            WorkTypeNP,
                            TGraph, TGraphDatum >

                            <<< grid_dims, block_dims, 0, stream.hip_stream >>> (

                            lwl_high->DeviceObject(), lwl_low->DeviceObject(),
                            rwl_in->DeviceObject(), rwl_out->DeviceObject(),
                            fused_chunk_size, global_prio,
                            high_work_counter, low_work_counter,
                            kernel_internal_counter, send_signal_ptr,
                            barrier_lifetime,
                            bfs::opt::SplitOps(m_graph, m_levels_datum),
                            m_graph, m_levels_datum
                            );
                    }
                    else
                    {
                        groute::FusedWork <
                            groute::RunNTimes<1>, local_work_t, remote_work_t, level_t, SplitOps,
                            WorkType,
                            TGraph, TGraphDatum >

                            << < grid_dims, block_dims, 0, stream.hip_stream >> > (

                            lwl_high->DeviceObject(), lwl_low->DeviceObject(),
                            rwl_in->DeviceObject(), rwl_out->DeviceObject(),
                            fused_chunk_size, global_prio,
                            high_work_counter, low_work_counter,
                            kernel_internal_counter, send_signal_ptr,
                            barrier_lifetime,
                            bfs::opt::SplitOps(m_graph, m_levels_datum),
                            m_graph, m_levels_datum
                            );
                    }
                }
            }
        };

        struct Algo
        {
            static const char* NameLower()      { return "bfs"; }
            static const char* Name()           { return "BFS"; }

            static void Init(
                groute::graphs::traversal::Context<bfs::opt::Algo>& context,
                groute::graphs::multi::CSRGraphAllocator& graph_manager,
                groute::router::Router<remote_work_t>& worklist_router,
                groute::opt::DistributedWorklist<local_work_t, remote_work_t, bfs::opt::SplitOps>& distributed_worklist)
            {
                index_t source_node = min(max(0, FLAGS_source_node), context.host_graph.nnodes - 1);

                auto partitioner = graph_manager.GetGraphPartitioner();
                if (partitioner->NeedsReverseLookup())
                {
                    source_node = partitioner->GetReverseLookupFunc()(source_node);
                }

                // Report the initial work
                distributed_worklist.ReportHighPrioWork(1, 0, "Host", groute::Device::Host, true);

                std::vector<remote_work_t> initial_work;
                initial_work.push_back(remote_work_t(source_node, 0));

                groute::router::ISender<remote_work_t>* work_sender = worklist_router.GetSender(groute::Device::Host);
                work_sender->Send(
                    groute::Segment<remote_work_t>(&initial_work[0], 1), groute::Event());
                work_sender->Shutdown();
            }

            template<typename TGraphAllocator, typename TGraphDatum, typename...UnusedData>
            static std::vector<level_t> Gather(TGraphAllocator& graph_allocator, TGraphDatum& levels_datum, UnusedData&... data)
            {
                graph_allocator.GatherDatum(levels_datum);
                return levels_datum.GetHostData();
            }

            template<typename...UnusedData>
            static std::vector<level_t> Host(groute::graphs::host::CSRGraph& graph, UnusedData&... data)
            {
                return BFSHost(graph, min(max(0, FLAGS_source_node), graph.nnodes - 1));
            }

            static int Output(const char *file, const std::vector<level_t>& levels)
            {
                return BFSOutput(file, levels);
            }

            static int CheckErrors(const std::vector<level_t>& levels, const std::vector<level_t>& regression)
            {
                return BFSCheckErrors(levels, regression);
            }
        };
    }
}

bool TestBFSAsyncMultiOptimized(int ngpus)
{
    typedef bfs::opt::FusedProblem<groute::graphs::dev::CSRGraphSeg, groute::graphs::dev::GraphDatum<level_t>> ProblemType;
    typedef groute::graphs::traversal::FusedSolver<
        bfs::opt::Algo, ProblemType, 
        bfs::opt::local_work_t , bfs::opt::remote_work_t, level_t, 
        bfs::opt::SplitOps, 
        groute::graphs::dev::CSRGraphSeg, groute::graphs::dev::GraphDatum<level_t>> SolverType;

    groute::graphs::traversal::__MultiRunner__Opt__ <
        bfs::opt::Algo,
        ProblemType,
        SolverType,
        bfs::opt::SplitOps,
        bfs::opt::local_work_t,
        bfs::opt::remote_work_t,
        groute::graphs::multi::NodeOutputGlobalDatum<level_t> > runner;

    groute::graphs::multi::NodeOutputGlobalDatum<level_t> levels_datum;
    
    bool retval = runner(ngpus, levels_datum);
    if(FLAGS_exitonerror && !retval)
        exit(100);
    return retval;
}

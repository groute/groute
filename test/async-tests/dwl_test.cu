#include "hip/hip_runtime.h"
// Groute: An Asynchronous Multi-GPU Programming Framework
// http://www.github.com/groute/groute
// Copyright (c) 2017, A. Barak
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// * Redistributions of source code must retain the above copyright notice,
//   this list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the names of the copyright holders nor the names of its 
//   contributors may be used to endorse or promote products derived from this
//   software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include <gtest/gtest.h>

#include "cuda_gtest_utils.h"
#include "test_common.h"

#include <vector>
#include <algorithm>
#include <thread>
#include <memory>

#include <groute/event_pool.h>
#include <groute/dwl/distributed_worklist.cuh>
#include <groute/dwl/workers.cuh>

#include <utils/cuda_utils.h>


namespace histogram
{
    struct CountWork
    {
        template<
            typename WorkSource, typename WorkTarget>
            __device__ static void work(
            const WorkSource& work_source, WorkTarget& work_target,
            int offset, int *bins
            )
        {
            uint32_t tid = TID_1D;
            uint32_t nthreads = TOTAL_THREADS_1D;

            uint32_t work_size = work_source.get_size();

            for (uint32_t i = 0 + tid; i < work_size; i += nthreads)
            {
                atomicAdd(bins + work_source.get_work(i) - offset, 1);
            }
        }
    };

    struct DWCallbacks
    {
        __device__ __forceinline__ groute::SplitFlags on_receive(int work)
        {
            return ((work / m_seg_size) == m_seg_index)
                ? groute::SF_Take
                : groute::SF_Pass;
        }

        __device__ __forceinline__ groute::SplitFlags on_send(int work)
        {
            return ((work / m_seg_size) == m_seg_index)
                ? groute::SF_Take
                : groute::SF_Pass;
        }
    
        __device__ __forceinline__ int pack(int work)
        {
            return work;
        }

        __device__ __forceinline__ int unpack(int work)
        {
            return work;
        }
        
        __device__ __forceinline__ bool should_defer(int work, const int& global_threshold)
        {
            return false;
        }
        
        __device__ __host__ DWCallbacks(int split_seg_index, int split_seg_size)
            : m_seg_index(split_seg_index), m_seg_size(split_seg_size)
        {
        }

        DWCallbacks() : m_seg_index(-1), m_seg_size(-1) { }

    private:
        int m_seg_index;
        int m_seg_size;
    };
}

template<typename TWorker>
void TestHistogramWorklist(int ngpus, size_t histo_size, size_t work_size)
{
    size_t histo_seg_size = histo_size / ngpus;
    histo_size = histo_seg_size * ngpus;

    ASSERT_GT(histo_seg_size, 0);

    size_t num_exch_buffs = 4 * ngpus;
    size_t exch_packet_size = work_size; 

    groute::Context context(ngpus);
    context.configuration.verbose = false;
    context.configuration.trace = false;

    groute::Endpoint host = groute::Endpoint::HostEndpoint(0);

    srand(static_cast <unsigned> (22522));
    std::vector<int> initial_work;
    for (size_t ii = 0, count = work_size; ii < count; ++ii)
    {
        initial_work.push_back((rand()*round_up(histo_size, RAND_MAX)) % histo_size);
    }

    std::vector<int*> dev_segs(ngpus);

    for (int i = 0; i < ngpus; ++i)
    {
        context.SetDevice(i);

        CUASSERT_NOERR(hipMalloc(&dev_segs[i], histo_seg_size * sizeof(int)));
        CUASSERT_NOERR(hipMemset(dev_segs[i], 0, histo_seg_size * sizeof(int)));
    }

    // Prepare DistributedWorklist parameters
    groute::EndpointList worker_endpoints = groute::Endpoint::Range(ngpus);
    std::map<groute::Endpoint, histogram::DWCallbacks> callbacks;
    for (int i = 0; i < ngpus; ++i)
    {
        callbacks[worker_endpoints[i]] = histogram::DWCallbacks(i, histo_seg_size);
    }

    groute::DistributedWorklist<int, int, histogram::DWCallbacks, TWorker> 
        distributed_worklist(context, { host }, worker_endpoints, callbacks, exch_packet_size, num_exch_buffs, 0);

    distributed_worklist.ReportInitialWork(initial_work.size(), host);
    distributed_worklist
        .GetLink(host)
        .Send(groute::Segment<int>(&initial_work[0], initial_work.size()), groute::Event());

    std::vector<std::thread> workers;
    groute::internal::Barrier barrier(ngpus);

    for (int i = 0; i < ngpus; ++i)
    {
        std::thread worker([&, i]()
        {
            context.SetDevice(i);
            groute::Stream stream = context.CreateStream(i);

            barrier.Sync();

            //
            // Start processing  
            //

            // Loop over the work until convergence  
            distributed_worklist.Work(i, stream, i*histo_seg_size, dev_segs[i]);

            stream.Sync();
        });

        workers.push_back(std::move(worker));
    }

    for (size_t i = 0; i < ngpus; ++i)
    {
        // Join workers  
        workers[i].join();
    }

    std::vector<int> regression_segs(histo_seg_size*ngpus, 0);
    std::vector<int> host_segs(histo_seg_size*ngpus);

    for (auto it : initial_work)
    {
        ++regression_segs[it];
    }

    for (int i = 0; i < ngpus; ++i)
    {
        context.SetDevice(i);
        CUASSERT_NOERR(hipMemcpy(&host_segs[i*histo_seg_size], dev_segs[i], histo_seg_size * sizeof(int), hipMemcpyDeviceToHost));
    }

    int over_errors = 0, miss_errors = 0;
    std::vector<int> over_error_indices, miss_error_indices;

    for (int i = 0; i < histo_size; ++i)
    {
        int hv = host_segs[i];
        int rv = regression_segs[i];

        if (hv > rv)
        {
            ++over_errors;
            over_error_indices.push_back(i);
        }

        else if (hv < rv)
        {
            ++miss_errors;
            miss_error_indices.push_back(i);
        }
    }

    ASSERT_EQ(0, over_errors + miss_errors);

    for (size_t i = 0; i < ngpus; ++i)
    {
        CUASSERT_NOERR(hipFree(dev_segs[i]));
    }
}

    
typedef groute::FusedWorker<true, int, int, int, histogram::DWCallbacks, histogram::CountWork, int, int*> FusedWorkerType;
typedef groute::Worker<int, int, histogram::DWCallbacks, histogram::CountWork, int, int*> WorkerType;


TEST(DWL, Worker_2)
{
    TestHistogramWorklist<WorkerType>(2, 1024, 4096);
    TestHistogramWorklist<WorkerType>(2, 1024, 20000);
    TestHistogramWorklist<WorkerType>(2, 10000, 4096);
    TestHistogramWorklist<WorkerType>(2, 10000, 200000);
}

TEST(DWL, Worker_4)
{
    TestHistogramWorklist<WorkerType>(4, 2048, 4096);
    TestHistogramWorklist<WorkerType>(4, 2048, 20000);
    TestHistogramWorklist<WorkerType>(4, 10000, 4096);
    TestHistogramWorklist<WorkerType>(4, 10000, 200000);
}

TEST(DWL, Worker_8)
{
    TestHistogramWorklist<WorkerType>(8, 1024, 4096);
    TestHistogramWorklist<WorkerType>(8, 1024, 20000);
    TestHistogramWorklist<WorkerType>(8, 10000, 4096);
    TestHistogramWorklist<WorkerType>(8, 10000, 200000);
}

TEST(DWL, Worker_N)
{
    TestHistogramWorklist<WorkerType>(3, 10000, 20000);
    TestHistogramWorklist<WorkerType>(4, 10000, 20000);
    TestHistogramWorklist<WorkerType>(5, 10000, 20000);
    TestHistogramWorklist<WorkerType>(15, 10000, 20000);
    TestHistogramWorklist<WorkerType>(27, 10000, 20000);
}


TEST(DWL, FusedWorker_2)
{
    TestHistogramWorklist<FusedWorkerType>(2, 1024, 4096);
    TestHistogramWorklist<FusedWorkerType>(2, 1024, 20000);
    TestHistogramWorklist<FusedWorkerType>(2, 10000, 4096);
    TestHistogramWorklist<FusedWorkerType>(2, 10000, 200000);
}

TEST(DWL, FusedWorker_4)
{
    TestHistogramWorklist<FusedWorkerType>(4, 2048, 4096);
    TestHistogramWorklist<FusedWorkerType>(4, 2048, 20000);
    TestHistogramWorklist<FusedWorkerType>(4, 10000, 4096);
    TestHistogramWorklist<FusedWorkerType>(4, 10000, 200000);
}

TEST(DWL, FusedWorker_8)
{
    TestHistogramWorklist<FusedWorkerType>(8, 1024, 4096);
    TestHistogramWorklist<FusedWorkerType>(8, 1024, 20000);
    TestHistogramWorklist<FusedWorkerType>(8, 10000, 4096);
    TestHistogramWorklist<FusedWorkerType>(8, 10000, 200000);
}

TEST(DWL, FusedWorker_N)
{
    TestHistogramWorklist<FusedWorkerType>(3, 10000, 20000);
    TestHistogramWorklist<FusedWorkerType>(4, 10000, 20000);
    TestHistogramWorklist<FusedWorkerType>(5, 10000, 20000);
    TestHistogramWorklist<FusedWorkerType>(15, 10000, 20000);
    TestHistogramWorklist<FusedWorkerType>(27, 10000, 20000);
}